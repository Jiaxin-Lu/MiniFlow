#include "hip/hip_runtime.h"
#include "./c_runtime_api.h"
#include <cassert>
#include <cstdio>
#include <hip/hip_runtime.h>

// y = inputs[0], y_ = inputs[1]
// np.mean(-np.sum(y_ * np.log(softmax(y)), axis=1), keepdims=True)
__global__ void matrix_softmax_cross_entropy_kernel(int nrow, int ncol,
                                                    const float *input_a,
                                                    const float *input_b,
                                                    float *output) {
  // Dynamic shared memory, size provided at kernel launch.
  extern __shared__ float loss_per_row[];
  // Two dimensional thread blocks.
  int y = blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x +
          threadIdx.x;
  if (y >= nrow) {
    return;
  }
  input_a += y * ncol;
  input_b += y * ncol;
  float maxval = *input_a;
  // Find max for a row.
  for (int x = 1; x < ncol; ++x) {
    maxval = max(maxval, input_a[x]);
  }
  // Deduct by max for a row, and raise to exp.
  float sum = 0;
  for (int x = 0; x < ncol; ++x) {
    sum += exp(input_a[x] - maxval);
  }
  // Compute per-row loss.
  float loss = 0;
  for (int x = 0; x < ncol; ++x) {
    loss -= input_b[x] * log(exp(input_a[x] - maxval) / sum);
  }
  loss_per_row[y] = loss;
  __syncthreads();
  // Compute reduce_mean across rows.
  float mean_loss = 0;
  // Use a single thread to reduce mean across rows.
  if ((threadIdx.x == 0) && (threadIdx.y == 0)) {
    for (int i = 0; i < nrow; ++i) {
      mean_loss += loss_per_row[i];
    }
    mean_loss /= nrow;
    output[0] = mean_loss;
  }
}
const int THREADS_PER_BLOCK = 512;
const int THREADS_PER_BLOCK_H = 256;
__global__ void array_set_kernel(float *A, float val, int n)
{
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < n)
  {
    A[i] = val;
  }
}
int DLGpuArraySet(DLArrayHandle arr, float value) { 
  int n = 1;
  for (int i=0;i<arr->ndim;++i) n *= arr->shape[i];
  float* input_val = (float *)arr->data;
  int nblocks = (n+THREADS_PER_BLOCK -1) / THREADS_PER_BLOCK;
  array_set_kernel<<<nblocks, THREADS_PER_BLOCK>>>(input_val, value, n);
  return 0;
}

__global__ void broadcast_to_kernel(const float* input, float *output, int in_n, int out_n)
{
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < out_n)
  {
    output[i] = input[i%in_n];
  }
}
int DLGpuBroadcastTo(const DLArrayHandle input, DLArrayHandle output) {
  int in_n = 1;
  for (int i=0;i<input->ndim;++i) in_n *= input->shape[i];
  int out_n = 1;
  for (int i=0;i<output->ndim;++i) out_n *= output->shape[i];
  const float* input_val = (const float*)input->data;
  float* output_val = (float*)output->data;
  int nblocks = (out_n+THREADS_PER_BLOCK_H -1) / THREADS_PER_BLOCK_H;
  broadcast_to_kernel<<<nblocks, THREADS_PER_BLOCK_H>>>(input_val, output_val, in_n, out_n);

  return 0;
}
__global__ void reducesum_axis_zero_kernel(const float* input, float *output, int m, int n)
{
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < n)
  {
    output[i] = 0;
    for (int k=0;k<m;++k) output[i] += input[i+n*k];
  }
}
int DLGpuReduceSumAxisZero(const DLArrayHandle input, DLArrayHandle output) {
  int axisa = input->shape[0];
  int axisb = 1;
  for (int i=1;i<input->ndim;++i) axisb *= input->shape[i];
  const float* input_val = (const float*)input->data;
  float* output_val = (float*) output->data;
  int nblocks = (axisb+THREADS_PER_BLOCK_H -1) / THREADS_PER_BLOCK_H;
  reducesum_axis_zero_kernel<<<nblocks, THREADS_PER_BLOCK_H>>>(input_val, output_val, axisa, axisb);

  return 0;
}
__global__ void matrix_elementwise_add_kernel(const float *A, const float *B, float *output, int n)
{
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < n)
  {
    output[i] = A[i] + B[i];
  }
}
int DLGpuMatrixElementwiseAdd(const DLArrayHandle matA,
                              const DLArrayHandle matB, DLArrayHandle output) {
  int n = 1;
  for (int i=0;i<matA->ndim;++i) n *= matA->shape[i];
  const float* matA_ = (const float*)matA->data;
  const float* matB_ = (const float*)matB->data;
  float *output_val = (float*) output->data;
  int nblocks = (n+THREADS_PER_BLOCK -1) / THREADS_PER_BLOCK;
  matrix_elementwise_add_kernel<<<nblocks, THREADS_PER_BLOCK>>>(matA_,matB_,output_val,n);
  return 0;
}
__global__ void matrix_elementwise_add_by_const_kernel(const float *A, const float val, float *output, int n)
{
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i<n)
  {
    output[i] = A[i] + val;
  }
}
int DLGpuMatrixElementwiseAddByConst(const DLArrayHandle input, float val,
                                     DLArrayHandle output) {
  int n = 1;
  for (int i=0;i<input->ndim;++i) n *= input->shape[i];
  const float* matA_ = (const float*)input->data;
  float *output_val = (float*) output->data;
  int nblocks = (n+THREADS_PER_BLOCK -1) / THREADS_PER_BLOCK;
  matrix_elementwise_add_by_const_kernel<<<nblocks, THREADS_PER_BLOCK>>>(matA_,val,output_val,n);
  return 0;
}

__global__  void matrix_elementwise_multiply_kernel(const float *A, const float *B, float *output, index_t n)
{
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < n)
  {
    output[i] = A[i]*B[i];
  }
}
int DLGpuMatrixElementwiseMultiply(const DLArrayHandle matA,
                                   const DLArrayHandle matB,
                                   DLArrayHandle output) {
  int n = 1;
  for (int i=0;i<matA->ndim;++i) n*=matA->shape[i];
  const float* mata_ = (const float*) matA->data;
  const float* matb_ = (const float*) matB->data;
  float *output_val = (float *)output->data;
  int nblocks = (n+THREADS_PER_BLOCK -1) / THREADS_PER_BLOCK;
  matrix_elementwise_multiply_kernel<<<nblocks, THREADS_PER_BLOCK>>>(mata_,matb_,output_val,n);
  return 0;
}

__global__ void matrix_elementwise_multiply_by_const_kernel(const float *A, const float val, float *output, int n)
{
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i<n)
  {
    output[i] = A[i] * val;
  }
}
int DLGpuMatrixMultiplyByConst(const DLArrayHandle input, float val,
                               DLArrayHandle output) {
  int n = 1;
  for (int i=0;i<input->ndim;++i) n *= input->shape[i];
  const float* matA_ = (const float*)input->data;
  float *output_val = (float*) output->data;
  int nblocks = (n+THREADS_PER_BLOCK -1) / THREADS_PER_BLOCK;
  matrix_elementwise_multiply_by_const_kernel<<<nblocks, THREADS_PER_BLOCK>>>(matA_,val,output_val,n);
  return 0;
}
#define BLOCK_SIZE 16
// void matmul_COPY(const float* A, float* B, int &x, int &y, bool trans)
// {
//   B = (float*)malloc(x*y*sizeof(float));
//   if (!trans)
//   {
//     for (int xx=0;xx<x;++xx)
//       for (int yy=0;yy<y;++yy)
//       {
//         *(B+xx*y+yy) = *(A+xx*y+yy);
//       }
//   }
//   else
//   {
//     for (int xx=0;xx<x;++xx)
//       for (int yy=0;yy<y;++yy)
//       {
//         *(B+yy*x+xx) = *(A+xx*y+yy);
//       }
//     int t = x;
//     x = y, y = t;
//   }
// }
__global__ void matmul_kernel(const float* A,const float *B, float *C,int ax,int ay,int bx, int by, bool TA, bool TB)
{
  float Cval = 0.0;
  int row = blockIdx.y*blockDim.y + threadIdx.y;
  int col = blockIdx.x*blockDim.x + threadIdx.x;
  if (row >= ax || col >= by) return;
  for (int i=0;i<ay;++i)
  {
    int ida = TA ? i*ax+row: row*ay+i;
    int idb = TB ? col*bx + i: i*by + col;
    Cval += A[ida] * B[idb];
  }
  C[row*by + col] = Cval;
}
inline void swap(int &x, int &y)
{
  int t = x;
  x = y, y = t;
}
int DLGpuMatrixMultiply(const DLArrayHandle matA, bool transposeA,
                        const DLArrayHandle matB, bool transposeB,
                        DLArrayHandle matC) {
  int lx = matA->shape[0], ly = matA->shape[1];
  int rx = matB->shape[0], ry = matB->shape[1];
  // float *lvh = (float *)malloc(lx*ly*sizeof(float));
  // float *rvh = (float *)malloc(rx*ry*sizeof(float));
  // const float* lvh_ = (const float*)matA->data;
  // const float* rvh_ = (const float*)matB->data;'
  const float *lvh = (const float*)matA->data;
  const float *rvh = (const float*)matB->data;
  float* output_val = (float*)matC->data;
  if (transposeA) swap(lx, ly);
  if (transposeB) swap(rx, ry);
  // matmul_COPY(lvh_, lvh, lx, ly, transposeA);
  // matmul_COPY(rvh_, rvh, rx, ry, transposeB);
  // puts("done");
  // int size = lx * ly * sizeof(float);
  // float* dA, dB;
  // hipError_t err = hipMalloc(&dA, size);
  // err = hipMemcpy(dA, lvh,size,hipMemcpyHostToDevice);
  // size = rx * ry * sizeof(float);
  // err = hipMalloc(&dB, size);
  // err = hipMemcpy(dB,rvh,size,hipMemcpyHostToDevice);
  // float* dC;
  // size = lx * ry * sizeof(float);
  // err = hipMalloc(&dC,size);

  dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
  dim3 dimGrid((ry + dimBlock.x - 1) / dimBlock.x, (lx + dimBlock.y - 1) / dimBlock.y);
  matmul_kernel<<<dimGrid, dimBlock>>>(lvh, rvh, output_val, lx, ly, rx, ry, transposeA, transposeB);
  // err = hipMemcpy(output_val, dC, size, hipMemcpyDeviceToHost);
  // hipFree(dA);
  // hipFree(dB);

//UNDONE!!!
  /* TODO: Your code here */
  // Hint: DO NOT use cublas
  return 0;
}
__global__ void relu_kernel(const float *A, float *output, int n)
{
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < n)
  {
    output[i] = (A[i] > 0) ? A[i]:0;
  }
}
int DLGpuRelu(const DLArrayHandle input, DLArrayHandle output) {
  int n = 1;
  for (int i=0;i<input->ndim;++i) n*=input->shape[i];
  const float* input_val = (const float*)input->data;
  float* output_val = (float*)output->data;
  int nblocks = (n+THREADS_PER_BLOCK -1) / THREADS_PER_BLOCK;
  relu_kernel<<<nblocks, THREADS_PER_BLOCK>>>(input_val, output_val, n);  
  return 0;
}

__global__ void relu_gradient_kernel(const float *A, const float *B, float *output, int n)
{
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < n)
  {
    output[i] = (A[i] > 0) ? B[i]:0;
  }
}
int DLGpuReluGradient(const DLArrayHandle input, const DLArrayHandle in_grad,
                      DLArrayHandle output) {
  int n = 1;
  for (int i=0;i<input->ndim;++i) n*=input->shape[i];
  const float* input_val = (const float*)input->data;
  const float* input_grad_val = (const float*)in_grad->data;
  float* output_val = (float*)output->data;
  int nblocks = (n+THREADS_PER_BLOCK -1) / THREADS_PER_BLOCK;
  relu_gradient_kernel<<<nblocks, THREADS_PER_BLOCK>>>(input_val, input_grad_val, output_val, n);
  return 0;
}

__global__ void softmax_kernel(const float* input, float* output, int nr, int nl)
{
  int i = blockDim.x * blockIdx.y * blockIdx.x + threadIdx.y * blockDim.x + threadIdx.x;
  if (i >= nr) return;
  input += i*nl;
  output += i*nl;
  float mx = *input;
  for (int x=1;x<nl;++x)
  {
    mx = max(mx, input[x]);
  }
  float sum = 0.0;
  for (int x=0;x<nl;++x)
  {
    sum += exp(input[x]-mx);
  }
  for (int x=0;x<nl;++x)
  {
    output[x] = exp(input[x]-mx)/sum;
  }
  
}
int DLGpuSoftmax(const DLArrayHandle input, DLArrayHandle output) {
  int nr = input->shape[0];
  int nl = input->shape[1];
  const float* input_val = (const float*)input->data;
  float* output_val = (float*)output->data;
  dim3 threads;
  if (nr < 1024)
  {
    threads.x = nr;
  }
  else
  {
    threads.x = 1024;
    threads.y = (nr + 1023)/1024;
  }
  softmax_kernel<<<1, threads, nr*sizeof(float)>>>(input_val, output_val, nr, nl);
  return 0;
}

int DLGpuSoftmaxCrossEntropy(const DLArrayHandle input_a,
                             const DLArrayHandle input_b,
                             DLArrayHandle output) {
  assert(input_a->ndim == 2);
  assert(input_b->ndim == 2);
  assert(output->ndim == 1);
  assert(input_a->shape[0] == input_b->shape[0] &&
         input_a->shape[1] == input_b->shape[1]);
  int nrow = input_a->shape[0];
  // Maximum x- or y-dimension of a block = 1024
  // But we need 'nrow' shared memory, and max shared memory is 48KB.
  // Conservatively allow max 16KB shared memory.
  assert(nrow <= 1024 * 4);
  int ncol = input_a->shape[1];
  const float *input_data_a = (const float *)input_a->data;
  const float *input_data_b = (const float *)input_b->data;
  float *output_data = (float *)output->data;
  dim3 threads;
  if (nrow <= 1024) {
    threads.x = nrow;
  } else {
    threads.x = 1024;
    threads.y = (nrow + 1023) / 1024;
  }
  // 1 block, each block with 'threads' number of threads with 'nrow' shared
  // memory size
  matrix_softmax_cross_entropy_kernel<<<1, threads, nrow * sizeof(float)>>>(
      nrow, ncol, input_data_a, input_data_b, output_data);
  return 0;
}